
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>
#include <math.h>
#include <sys/time.h>
#define SWAP(a,b) tempr=a;a=b;b=tempr
#define NUM_ITERACOES 1
#define ISIGN 1


//dados globais usados pelas threads
typedef struct strDadosThread
{
    unsigned long m;
    unsigned long mmax;
    unsigned long istep;
    double wr;
    double wi; 
    unsigned long tamBloco;
}DadosThread;

//estrutura de dados para uso da instrucao rdtsc (contador de timestamp em clocks nivel HW)
typedef union {
    unsigned long long int64;
    struct {
        unsigned int lo, hi;
    } int32;
} tsc_counter;

//macro para uso da instrucao rdtsc: RDTSC retorna o TSC (Time Stamp Counter) (numero de ciclos desde o ultimo reset) 
//nos registradores EDX:EAX
#define RDTSC(cpu_c)                 \
__asm__ __volatile__ ("rdtsc" :    \
"=a" ((cpu_c).int32.lo),           \
"=d" ((cpu_c).int32.hi) )


//dados globais
float *data, *cudaData;
DadosThread *dadosThreads, *cudaDadosThreads;
int qtdThreads, qtdBlocos;
unsigned long qtdElementos, tamArray;


void imprimeVetor(){
    int posicao;
    for(posicao=1;posicao<tamArray;posicao++){
        printf("%2.2f ",data[posicao]);
    }
    printf("\n\n");
}

void ordenaBitReverso(){
    
    unsigned long i,j,m;
    float tempr;
    j=1;
    
    for (i=1;i<tamArray;i+=2) {
        if (j > i) {
            SWAP(data[j],data[i]);
            SWAP(data[j+1],data[i+1]);
        }
        m=qtdElementos;
        while (m >= 2 && j > m) {
            j -= m;
            m >>= 1;
        }
        j += m;
    }
}

void inicializaArray(){
    
    unsigned long i;
    
    for (i=1; i<tamArray; i+=8) {
        data[i]=0.;
        data[i+1]=0.;
        data[i+2]=1.;
        data[i+3]=0.;
        data[i+4]=2.;
        data[i+5]=0.;
        data[i+6]=3.;
        data[i+7]=0.;
    }
    
}

void calculoButterflyBloco(DadosThread *dthread){

	//TODO: calcular o valor de M baseado no ID da thread
	unsigned long m = dthread->m;

	unsigned long mmax = dthread->mmax; 
	unsigned long istep = dthread->istep;
	unsigned long tamBloco = dthread->tamBloco; 
	double wr = dthread->wr;
	double wi = dthread->wi;
	float tempr,tempi;
	unsigned long j,i,bloco;
	bloco=1;

	for (i=m;bloco<=tamBloco;i+=istep) {
		j=i+mmax;                                
		//printf("          >>> FOR 2 -> wi:%f wr:%f istep:%lu mmax:%lu i:%lu j:%lu\n",wi,wr,istep, mmax, i, j);
		tempr=wr*data[j]-wi*data[j+1];
		tempi=wr*data[j+1]+wi*data[j];
		data[j]=data[i]-tempr;
		data[j+1]=data[i+1]-tempi;
		data[i] += tempr;
		data[i+1] += tempi; 
		bloco++;
	}
}

int fftCuda(){
    
   //aloca espaco para os vetores na GPU
   if (hipMalloc (&cudaData, sizeof(float) * tamArray) != hipSuccess) 
        { printf("Erro hipMalloc do array 'data'\n"); return -1; }

    //Copia os dados da memoria principal para a memoria do dispositivo
   if (hipMemcpy(cudaData, data, sizeof(float) * tamArray, hipMemcpyHostToDevice) != hipSuccess) 
        { printf("Erro hipMemcpy do array 'data'\n"); return -1; }

   //aloca espaco para os dados usados no algoritmo na GPU
   if (hipMalloc (&cudaDadosThreads, sizeof(DadosThread)) != hipSuccess) 
        { printf("Erro hipMalloc do struct 'DadosThread'\n"); return -1; }
    
     unsigned long mmax,m,istep;
     double wtemp,wr,wpr,wpi,wi,theta;

        mmax=2;
        while (tamArray > mmax) {
        //printf(">>> Inicio do while -> mmax:%lu\n",mmax);
                istep=mmax << 1;
                theta=ISIGN*(6.28318530717959/mmax);
                wtemp=sin(0.5*theta);
                wpr = -2.0*wtemp*wtemp;
                wpi=sin(theta);
                wr=1.0;
                wi=0.0;
                for (m=1;m<mmax;m+=2) {
                //printf("     >>> FOR 1 -> m:%lu mmax:%lu\n",m,mmax); 

			unsigned long tamBloco = qtdElementos/mmax/qtdThreads;
                                     
			//preeenche os dados para enviar para a thread
			dadosThreads->m = m;
			dadosThreads->mmax = mmax;
			dadosThreads->istep = istep;
			dadosThreads->wr = wr;
			dadosThreads->wi = wi; 
			dadosThreads->tamBloco = tamBloco;

			//Copia os dados do execucao para a memoria do dispositivo
			if (hipMemcpy(cudaDadosThreads, dadosThreads, sizeof(DadosThread), hipMemcpyHostToDevice) != hipSuccess) 
				{ printf("Erro hipMemcpy do array 'data'\n"); return -1; }

			//executa o kernel

			wr=(wtemp=wr)*wpr-wi*wpi+wr;
			wi=wi*wpr+wtemp*wpi+wi;
                }
		//obtem o istep da GPU
		if (hipMemcpy(dadosThreads, cudaDadosThreads, sizeof(unsigned long), hipMemcpyDeviceToHost) != hipSuccess) 
		{ printf("Erro cudaMemcpy\n"); return -1; }

		//e atualiza os valores internos para a proxima rodada
                mmax=dadosThreads->istep;
        //printf("<<< Final do while -> istep:%lu mmax:%lu\n\n",istep,mmax);
        }


	//Copia os dados da memoria do dispositivo para a memoria principal
	if (hipMemcpy(data, cudaData, sizeof(float) * tamArray, hipMemcpyDeviceToHost) != hipSuccess) 
        	{ printf("Erro hipMemcpy do array 'data'\n"); return -1; }
	
	//retorna sucesso
	return 0;

}

int main(int argc, char *argv[]) {

	struct timeval inicio, fim;
	tsc_counter tsc1, tsc2;
	long long unsigned int clock;
	double tempo;


	if(argc < 4) {
	        printf("ERRO: informe s quantidade de elementos do FFT, a quantidade de threads e a quantidade de blocos :: fft-cuda <qtdElementos> <qtdThreads> <qtdBlocos>\n");
	        return -1;
	}
    
	//inicializando valores do algoritmo
	qtdElementos = strtoul(argv[1],NULL,10);
	tamArray = (qtdElementos * 2) + 1;
	qtdThreads = atoi(argv[2]);
	qtdBlocos = atoi(argv[3]);
    
	//printf("sera processada uma fft com Elementos:%lu, Threads:%d, PesoThread:%d\n",qtdElementos,qtdThreads,quantidadeBlocos);

	//alocando memoria para o vetor de dados
	data = (float*)malloc(sizeof(float) * tamArray);
	if(data==NULL) { printf("Erro malloc vetor 'data'\n"); return(-1); }

	//alocando memoria para o struct dos dados da thread
	dadosThreads = (DadosThread*)malloc(sizeof(DadosThread));
	if(data==NULL) { printf("Erro malloc no struct 'DadosThread'\n"); return(-1); }


	//warmup
	//inicializaArray();
	//ordenaBitReverso();
	//fftCuda();

	//repete as execucoes para calcular a media
	unsigned long count;
    
	//mede o tempo de execução da funcao (media de NUM_ITERACOES repeticoes) 
	gettimeofday(&inicio, NULL);
	RDTSC(tsc1);
	for(count=0;count<NUM_ITERACOES;count++){
        
	        inicializaArray();
	        //imprimeVetor();
        
	        ordenaBitReverso();
	        //imprimeVetor();
        
	        fftCuda();
	}

	gettimeofday(&fim, NULL);
	RDTSC(tsc2);
	tempo = (fim.tv_sec - inicio.tv_sec)*1000 + (fim.tv_usec - inicio.tv_usec)/1000; //calcula tempo em milisegundos
	clock = tsc2.int64 - tsc1.int64; //calcula numero de ciclos de CPU gastos
	printf("%lu\t%d\t%d\t%.1lf\t%.2e\t%.2e\n",qtdElementos,qtdThreads,qtdBlocos,tempo/NUM_ITERACOES,(double)clock/NUM_ITERACOES,clock/tempo);

	imprimeVetor();

	//liberando a memoria
	free(data);

	return 0;
}

