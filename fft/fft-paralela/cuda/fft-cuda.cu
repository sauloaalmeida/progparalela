
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>
#include <math.h>
#include <sys/time.h>
#define SWAP(a,b) tempr=a;a=b;b=tempr
#define NUM_ITERACOES 1
#define ISIGN 1


//dados globais usados pelas threads
typedef struct strDadosThread
{
    
    unsigned long tamArray;
    int qtdThreads;
    unsigned long m;
    unsigned long mmax;
    unsigned long istep;
    float wr;
    float wi; 
    unsigned long tamBloco;
}DadosThread;

//estrutura de dados para uso da instrucao rdtsc (contador de timestamp em clocks nivel HW)
typedef union {
    unsigned long long int64;
    struct {
        unsigned int lo, hi;
    } int32;
} tsc_counter;

//macro para uso da instrucao rdtsc: RDTSC retorna o TSC (Time Stamp Counter) (numero de ciclos desde o ultimo reset) 
//nos registradores EDX:EAX
#define RDTSC(cpu_c)                 \
__asm__ __volatile__ ("rdtsc" :    \
"=a" ((cpu_c).int32.lo),           \
"=d" ((cpu_c).int32.hi) )


//dados globais
float *data, *cudaData;
DadosThread *dadosThreads, *cudaDadosThreads;
int qtdThreads, qtdThreadsBlocos, pesoThreads;
unsigned long qtdElementos, tamArray;


void imprimeVetor(){
    int posicao;
    for(posicao=1;posicao<tamArray;posicao++){
        printf("%2.2f ",data[posicao]);
    }
    printf("\n\n");
}

void ordenaBitReverso(){
    
    unsigned long i,j,m;
    float tempr;
    j=1;
    
    for (i=1;i<tamArray;i+=2) {
        if (j > i) {
            SWAP(data[j],data[i]);
            SWAP(data[j+1],data[i+1]);
        }
        m=qtdElementos;
        while (m >= 2 && j > m) {
            j -= m;
            m >>= 1;
        }
        j += m;
    }
}

void inicializaArray(){
    
    unsigned long i;
    
    for (i=1; i<tamArray; i+=8) {
        data[i]=0.;
        data[i+1]=0.;
        data[i+2]=1.;
        data[i+3]=0.;
        data[i+4]=2.;
        data[i+5]=0.;
        data[i+6]=3.;
        data[i+7]=0.;
    }
    
}

void calculoButterflyBlocoLocal(float *data, DadosThread *dthread){

	unsigned long m = dthread->m;
	unsigned long mmax = dthread->mmax; 
	unsigned long istep = dthread->istep;
	unsigned long tamBloco = dthread->tamBloco; 
	float wr = dthread->wr;
	float wi = dthread->wi;
	float tempr,tempi;
	unsigned long j,i,bloco;
	bloco=1;

	for (i=m;bloco<=tamBloco;i+=istep) {
		j=i+mmax;                                
		//printf("          >>> FOR 2 -> wi:%f wr:%f istep:%lu mmax:%lu i:%lu j:%lu\n",wi,wr,istep, mmax, i, j);
		tempr=wr*data[j]-wi*data[j+1];
		tempi=wr*data[j+1]+wi*data[j];
		data[j]=data[i]-tempr;
		data[j+1]=data[i+1]-tempi;
		data[i] += tempr;
		data[i+1] += tempi; 
		bloco++;
	}
}


__global__ void calculoButterflyBlocoCuda(float *data, DadosThread *dthread){

	int threadID = blockIdx.x * blockDim.x + threadIdx.x;

	unsigned long m = (((dthread->tamArray - 1)/dthread->qtdThreads)*threadID)+dthread->m;
	unsigned long mmax = dthread->mmax; 
	unsigned long istep = dthread->istep;
	unsigned long tamBloco = dthread->tamBloco; 
	float wr = dthread->wr;
	float wi = dthread->wi;
	float tempr,tempi;
	unsigned long j,i,bloco;
	bloco=1;

	for (i=m;bloco<=tamBloco;i+=istep) {
		j=i+mmax;                                
		//printf("          >>> FOR 2 -> wi:%f wr:%f istep:%lu mmax:%lu i:%lu j:%lu\n",wi,wr,istep, mmax, i, j);
		tempr=wr*data[j]-wi*data[j+1];
		tempi=wr*data[j+1]+wi*data[j];
		data[j]=data[i]-tempr;
		data[j+1]=data[i+1]-tempi;
		data[i] += tempr;
		data[i+1] += tempi; 
		bloco++;
	}
}

int fftCuda(){


	unsigned long mmax,m,istep;
	unsigned long tamLoop,tamBloco;
	float wtemp,wr,wpr,wpi,wi,theta;
	mmax=2;
	tamLoop = qtdElementos/mmax;
	tamBloco = tamLoop/qtdThreads;
	int processaGPU = 0; 	
	int retornouDadosGPU = 0; 
	unsigned long totalPeso = (pow(qtdThreads,pesoThreads));
	

	if(qtdThreads > 1 && tamLoop >= totalPeso){

		processaGPU = 1;

		//aloca espaco para os vetores na GPU
		if (hipMalloc (&cudaData, sizeof(float) * tamArray) != hipSuccess) 
		{ printf("Erro hipMalloc do array 'data'\n"); return -1; }

		//Copia os dados da memoria principal para a memoria do dispositivo
		if (hipMemcpy(cudaData, data, sizeof(float) * tamArray, hipMemcpyHostToDevice) != hipSuccess) 
		{ printf("Erro hipMemcpy do array 'data'\n"); return -1; }

		//aloca espaco para os dados usados no algoritmo na GPU
		if (hipMalloc (&cudaDadosThreads, sizeof(DadosThread)) != hipSuccess) 
		{ printf("Erro hipMalloc do struct 'DadosThread'\n"); return -1; }	

	}
	
   
        while (qtdElementos >=  mmax) {
        //printf(">>> Inicio do while -> mmax:%lu\n",mmax);
                istep=mmax << 1;
                theta=ISIGN*(6.28318530717959/mmax);
                wtemp=sin(0.5*theta);
                wpr = -2.0*wtemp*wtemp;
                wpi=sin(theta);
                wr=1.0;
                wi=0.0;
                for (m=1;m<mmax;m+=2) {
                //printf("     >>> FOR 1 -> m:%lu mmax:%lu\n",m,mmax); 

			tamLoop = qtdElementos/mmax;
			tamBloco = tamLoop/qtdThreads;

			if(tamLoop < totalPeso && qtdThreads > 1){
				processaGPU = 0;
				if(!retornouDadosGPU){
					if (hipMemcpy(data, cudaData, sizeof(float) * tamArray, hipMemcpyDeviceToHost) != hipSuccess){ 
						printf("Erro hipMemcpy array 'data'\n"); return -1; 
					}
					retornouDadosGPU = 1;
				}			
			}
                                     
			//preeenche os dados do processamento
			dadosThreads->tamArray = tamArray;
    			dadosThreads->qtdThreads = qtdThreads;
			dadosThreads->m = m;
			dadosThreads->mmax = mmax;
			dadosThreads->istep = istep;
			dadosThreads->wr = wr;
			dadosThreads->wi = wi; 
			dadosThreads->tamBloco = tamBloco;

			if(processaGPU){
				
				//printf("vai processar na GPU tamLoop:%lu tamBloco:%lu\n",tamLoop,tamBloco);

				dadosThreads->tamBloco = tamBloco;

				//Copia os dados do execucao para a memoria do dispositivo
				if (hipMemcpy(cudaDadosThreads, dadosThreads, sizeof(DadosThread), hipMemcpyHostToDevice) != hipSuccess) 
					{ printf("Erro hipMemcpy do array 'data'\n"); return -1; }

				//executa o kernel
				calculoButterflyBlocoCuda<<<qtdThreads/qtdThreadsBlocos,qtdThreadsBlocos>>>(cudaData,cudaDadosThreads);
			}else{
				//printf("vai processar localmente tamLoop:%lu\n",tamLoop);
				//senao calcula localmente
				dadosThreads->tamBloco = tamLoop;
				calculoButterflyBlocoLocal(data,dadosThreads);
			}

			wr=(wtemp=wr)*wpr-wi*wpi+wr;
			wi=wi*wpr+wtemp*wpi+wi;
                }

		/*if(processaGPU){
			//obtem o istep da GPU
			if (cudaMemcpy(dadosThreads, cudaDadosThreads, sizeof(unsigned long), cudaMemcpyDeviceToHost) != cudaSuccess) { 
				printf("Erro cudaMemcpy do 'cudaDadosThreads'\n"); return -1; 
			}
		}*/

		//e atualiza os valores internos para a proxima rodada
                mmax=istep;
        //printf("<<< Final do while -> istep:%lu mmax:%lu\n\n",istep,mmax);
        }

	//libera a memoria na GPU
	if (hipFree(cudaData) != hipSuccess) { printf("Erro hipFree 'cudaData'\n"); return -1; }
	if (hipFree(cudaDadosThreads) != hipSuccess) { printf("Erro hipFree 'cudaDadosThreads'\n"); return -1; }
	
	//retorna sucesso
	return 0;

}

int main(int argc, char *argv[]) {

	struct timeval inicio, fim;
	tsc_counter tsc1, tsc2;
	long long unsigned int clock;
	float tempo;

	if(argc < 5) {
	        printf("ERRO: informe s quantidade de elementos do FFT, a quantidade de threads e a quantidade de threads por blocos e o peso das threads:: fft-cuda <qtdElementos> <qtdThreads> <qtdThreadsPorBloco> <pesoThreads>\n");
	        return -1;
	}
    
	//inicializando valores do algoritmo
	qtdElementos = strtoul(argv[1],NULL,10);
	tamArray = (qtdElementos * 2) + 1;
	qtdThreads = atoi(argv[2]);
	qtdThreadsBlocos = atoi(argv[3]);
	pesoThreads = atoi(argv[4]);
    
	//printf("sera processada uma fft com Elementos:%lu, Threads:%d, PesoThread:%d\n",qtdElementos,qtdThreads,quantidadeBlocos);

	//alocando memoria para o vetor de dados
	data = (float*)malloc(sizeof(float) * tamArray);
	if(data==NULL) { printf("Erro malloc vetor 'data'\n"); return(-1); }

	//alocando memoria para o struct dos dados da thread
	dadosThreads = (DadosThread*)malloc(sizeof(DadosThread));
	if(data==NULL) { printf("Erro malloc no struct 'DadosThread'\n"); return(-1); }


	//warmup
	//inicializaArray();
	//ordenaBitReverso();
	//fftCuda();

	//repete as execucoes para calcular a media
	unsigned long count;
    
	//mede o tempo de execução da funcao (media de NUM_ITERACOES repeticoes) 
	gettimeofday(&inicio, NULL);
	RDTSC(tsc1);
	for(count=0;count<NUM_ITERACOES;count++){
        
	        inicializaArray();
	        //imprimeVetor();
        
	        ordenaBitReverso();
	        //imprimeVetor();
        
	        fftCuda();
	}

	gettimeofday(&fim, NULL);
	RDTSC(tsc2);
	tempo = (fim.tv_sec - inicio.tv_sec)*1000 + (fim.tv_usec - inicio.tv_usec)/1000; //calcula tempo em milisegundos
	clock = tsc2.int64 - tsc1.int64; //calcula numero de ciclos de CPU gastos
	printf("%lu\t%d\t%d\t%d\t%.1lf\t%.2e\t%.2e\n",qtdElementos,qtdThreads,qtdThreadsBlocos,pesoThreads,tempo/NUM_ITERACOES,(float)clock/NUM_ITERACOES,clock/tempo);

	//imprimeVetor();

	//liberando a memoria
	free(data);
	free(dadosThreads);

	return 0;
}

