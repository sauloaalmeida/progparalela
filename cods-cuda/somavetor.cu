/* Disciplina: Ambientes de Programacao Paralela (MAB114) */
/* Prof.: Silvana Rossetto */
/* Codigo: Soma de vetores em CUDA */

/* Para compilar: nvcc -o somavetor somavetor.cu */


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <sys/time.h>

#define TAMANHO 10000 
//#define PRINT_XY //imprime os vetores de entrada
#define PRINT //imprime o vetor de saida

//funcao para execucao sequencial
void calc_seq(int n, float alpha, float *X, float *Y) {
   int i;
   for(i=0; i<n; i++)
       Y[i] = alpha * X[i] + Y[i];
}

//Kernel para execucao paralela em CUDA
__global__ void calc_par(int n, float alpha, float *X, float *Y) {
   int i = blockIdx.x * blockDim.x + threadIdx.x;
   if(i < n) 
      Y[i] = alpha * X[i] + Y[i];
}

//funcao para imprimir os elementos de um vetor de inteiros
void printVetor(float *vetor, int n) {
   for (int i=0; i<n; i++) 
      printf("%.1f ", vetor[i]);
   printf("\n");
   printf("\n");
   printf("\n");
}

//funcao para computar um intervalo de tempo em milisegundos
double calculaTempo(struct timeval startTime, struct timeval endTime) {
   unsigned int totalMicroSecs;
   double t;
   totalMicroSecs = (unsigned long long) (endTime.tv_sec - startTime.tv_sec) * 1000000 +
            (unsigned long long) (endTime.tv_usec - startTime.tv_usec);
   t = (double) totalMicroSecs/1000;
   return t;
}

//funcao principal
int main(int argc, char** argv) {
   float *y, *cudaY;
   float *x, *cudaX;
   float alpha;
   struct timeval inicio, fim;
   double tempo_seq, tempo_par;

   if(argc != 2) {
      puts("Erro: digite o nome do programa e o valor do coeficiente alpha");
      return -1;
   }
   //armazena parametro de entrada
   alpha = atof(argv[1]);

   //aloca memoria na CPU
   x = (float*) malloc (sizeof(float) * TAMANHO);
   if(x==NULL) { printf("Erro malloc vetor X\n"); return(-1); }
   y = (float*) malloc (sizeof(float) * TAMANHO);
   if(y==NULL) { printf("Erro malloc vetor Y\n"); return(-1); }

   //inicializa os dois vetores de entrada
   for(int i=0; i<TAMANHO; i++){
      x[i] = (float) i;
      y[i] = (float) i;
   }

   //imprime os vetores de entrada
   #ifdef PRINT_XY 
   printf("Vetor X\n"); 
   printVetor(x, TAMANHO);
   printf("Vetor Y\n");
   printVetor(y, TAMANHO);
   #endif

   //!!! ------------------------ executa sequencial ---------------------------------- !!!//
   gettimeofday(&inicio, NULL);
   calc_seq(TAMANHO, alpha, x, y);
   gettimeofday(&fim, NULL);

   tempo_seq = calculaTempo(inicio, fim); // em milisegundos
   #ifdef PRINT 
   printf("Vetor Y(calculo sequencial)\n");
   printVetor(y, TAMANHO);
   #endif


   //!!! ------------------------ executa em paralelo em CUDA -------------------------- !!!//
   //gettimeofday(&inicio, NULL);

   //inicializa os dois vetores de entrada
   for(int i=0; i<TAMANHO; i++){
      x[i] = (float) i;
      y[i] = (float) i;
   }
   //aloca espaco para os vetores na GPU
   if (hipMalloc (&cudaX, sizeof(float) * TAMANHO) != hipSuccess) 
        { printf("Erro cudaMalloc\n"); return -1; }
   if (hipMalloc (&cudaY, sizeof(float) * TAMANHO) != hipSuccess) 
        { printf("Erro cudaMalloc\n"); return -1; }

   //copia os vetores de entrada da CPU para a GPU
   if (hipMemcpy(cudaX, x, sizeof(float) * TAMANHO, hipMemcpyHostToDevice) != hipSuccess) 
       { printf("Erro cudaMemcpy\n"); return -1; }
   if (hipMemcpy(cudaY, y, sizeof(float) * TAMANHO, hipMemcpyHostToDevice) != hipSuccess) 
       { printf("Erro cudaMemcpy\n"); return -1; }

   //dispara o kernel paralelo
   gettimeofday(&inicio, NULL);
   int nblocks = (TAMANHO+255) / 256;
   calc_par <<<nblocks,256>>> (TAMANHO, alpha, cudaX, cudaY);
   gettimeofday(&fim, NULL);

   //copia resultado da GPU para a CPU
   if (hipMemcpy(y, cudaY, sizeof(float) * TAMANHO, hipMemcpyDeviceToHost) != hipSuccess) 
       { printf("Erro cudaMemcpy\n"); return -1; }
   
   //libera a memoria na GPU
   if (hipFree(cudaX) != hipSuccess) { printf("Erro cudaFree\n"); return -1; }
   if (hipFree(cudaY) != hipSuccess) { printf("Erro cudaFree\n"); return -1; }

   //gettimeofday(&fim, NULL);
   tempo_par = calculaTempo(inicio, fim); // em milisegundos
   #ifdef PRINT 
   printf("Vetor Y(calculo paralelo)\n");
   printVetor(y, TAMANHO);
   #endif

   //libera a memoria na CPU
   free(x);
   free(y);

   //------------------------------- imprime dos tempos de execucao ----------------------//
   printf("Tempo sequencial = %g mseg \n", tempo_seq);
   printf("Tempo paralelo = %g mseg \n", tempo_par);

   return 0;
}
