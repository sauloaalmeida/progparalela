/* Disciplina: Ambientes de Programacao Paralela (MAB114) */
/* Prof.: Silvana Rossetto */
/* Codigo: Soma de prefixos em CUDA */

/* Para compilar: nvcc -o prefixo prefixo.cu */


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <sys/time.h>

#define TAMANHO 65536
//#define PRINT

//funcao para execucao sequencial
void soma_prefixo_seq(int n, float *X) {
   int i;
   for(i=1; i<n; i++)
       X[i] = X[i-1] + X[i];
}

//Kernel para execucao paralela em CUDA (!!assume-se que o tamanho do vetor sera sempre potencia de 2!!)
__global__ void soma_prefixo_par(float *X) {
   int i = threadIdx.x;
   int n = blockDim.x;
   int offset;

   for(offset=1; offset<n; offset*=2) {
     int aux;
     if(i >= offset) { aux = X[i-offset]; }
     __syncthreads();
     if(i >= offset) { X[i] = aux + X[i]; }
     __syncthreads();
   }
}

//funcao para imprimir os elementos de um vetor de inteiros
void printVetor(float *vetor, int n) {
   for (int i=0; i<n; i++) 
      printf("%2.1f ", vetor[i]);
   printf("\n");
}

//funcao para computar um intervalo de tempo em milisegundos
double calculaTempo(struct timeval startTime, struct timeval endTime) {
   unsigned int totalMicroSecs;
   double t;
   totalMicroSecs = (unsigned long long) (endTime.tv_sec - startTime.tv_sec) * 1000000 +
            (unsigned long long) (endTime.tv_usec - startTime.tv_usec);
   t = (double) totalMicroSecs/1000;
   return t;
}

//funcao principal
int main(int argc, char** argv) {
   float *x, *cudaX;
   struct timeval inicio, fim;
   double tempo_seq, tempo_par;

   //aloca memoria na CPU
   x = (float*) malloc (sizeof(float) * TAMANHO);
   if(x==NULL) { printf("Erro malloc vetor X\n"); return(-1); }

   //inicialize o vetor de entrada
   for(int i=0; i<TAMANHO; i++){
      x[i] = (float) i;
   }

   //imprime o vetor de entrada
   #ifdef PRINT 
   printf("Vetor X\n"); 
   printVetor(x, TAMANHO);
   #endif

   //!!! ------------------------ executa sequencial ---------------------------------- !!!//
   gettimeofday(&inicio, NULL);
   soma_prefixo_seq(TAMANHO, x);
   gettimeofday(&fim, NULL);

   tempo_seq = calculaTempo(inicio, fim); // em milisegundos
   #ifdef PRINT 
   printf("Vetor X (calculo sequencial)\n");
   printVetor(x, TAMANHO);
   #endif

   //!!! ------------------------ executa em paralelo em CUDA -------------------------- !!!//
   //gettimeofday(&inicio, NULL);

   //aloca espaco para os vetores na GPU
   if (hipMalloc (&cudaX, sizeof(float) * TAMANHO) != hipSuccess) 
        { printf("Erro cudaMalloc\n"); return -1; }

   //copia os vetores de entrada da CPU para a GPU
   if (hipMemcpy(cudaX, x, sizeof(float) * TAMANHO, hipMemcpyHostToDevice) != hipSuccess) 
        { printf("Erro cudaMemcpy\n"); return -1; }

   //dispara o kernel paralelo
   gettimeofday(&inicio, NULL);
   int nblocks = 1;  
   //!!!ATENCAO: pode causar erro se o tamanho do vetor extrapolar o numero maximo de threads por bloco!!!
   soma_prefixo_par <<<nblocks,TAMANHO>>> (cudaX);
   gettimeofday(&fim, NULL);

   //copia resultado da GPU para a CPU
   if (hipMemcpy(x, cudaX, sizeof(float) * TAMANHO, hipMemcpyDeviceToHost) != hipSuccess) 
      { printf("Erro cudaMemcpy\n"); return -1; }
   
   //libera a memoria na GPU
   if (hipFree(cudaX) != hipSuccess) { printf("Erro cudaFree\n"); return -1; }

   //gettimeofday(&fim, NULL);
   tempo_par = calculaTempo(inicio, fim); // em milisegundos
   #ifdef PRINT 
   printf("Vetor X(calculo paralelo)\n");
   printVetor(x, TAMANHO);
   #endif
   
   //libera a memoria na CPU
   free(x);

   //------------------------------- imprime dos tempos de execucao ----------------------//
   printf("Tempo sequencial = %g mseg \n", tempo_seq);
   printf("Tempo paralelo = %g mseg \n", tempo_par);

   return 0;
}
